#include "hip/hip_runtime.h"
// under review by hwb, 3.4.2016

#include "device.hpp"

const float sigma_color = 30;     //in mm
const float sigma_space = 4.5;     // in pixels

__global__ void
bilateralKernel (const PtrStepSz<unsigned short> src,
                 PtrStep<unsigned short> dst,
                 float sigma_space2_inv_half, float sigma_color2_inv_half)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= src.cols || y >= src.rows)
    return;

  const int R = 6;       //static_cast<int>(sigma_space * 1.5);
  const int D = R * 2 + 1;

  int value = src.ptr (y)[x];

  int tx = min (x - D / 2 + D, src.cols - 1);
  int ty = min (y - D / 2 + D, src.rows - 1);

  float sum1 = 0;
  float sum2 = 0;

  for (int cy = max (y - D / 2, 0); cy < ty; ++cy)
  {
    for (int cx = max (x - D / 2, 0); cx < tx; ++cx)
    {
      int tmp = src.ptr (cy)[cx];

      float space2 = (x - cx) * (x - cx) + (y - cy) * (y - cy);
      float color2 = (value - tmp) * (value - tmp);

      float weight = __expf (-(space2 * sigma_space2_inv_half + color2 * sigma_color2_inv_half));

      sum1 += tmp * weight;
      sum2 += weight;
    }
  }

  int res = __float2int_rn (sum1 / sum2);
  dst.ptr (y)[x] = max (0, min (res, numeric_limits<short>::max ()));
}

__global__ void
pyrDownGaussKernel (const PtrStepSz<unsigned short> src, PtrStepSz<unsigned short> dst, float sigma_color)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  const int D = 5;

  int center = src.ptr (2 * y)[2 * x];

  int x_mi = max(0, 2*x - D/2) - 2*x;
  int y_mi = max(0, 2*y - D/2) - 2*y;

  int x_ma = min(src.cols, 2*x -D/2+D) - 2*x;
  int y_ma = min(src.rows, 2*y -D/2+D) - 2*y;

  float sum = 0;
  float wall = 0;

  float weights[] = {0.375f, 0.25f, 0.0625f} ;

  for(int yi = y_mi; yi < y_ma; ++yi)
      for(int xi = x_mi; xi < x_ma; ++xi)
      {
          int val = src.ptr (2*y + yi)[2*x + xi];

          if (abs (val - center) < 3 * sigma_color)
          {
            sum += val * weights[abs(xi)] * weights[abs(yi)];
            wall += weights[abs(xi)] * weights[abs(yi)];
          }
      }  dst.ptr (y)[x] = static_cast<int>(sum /wall);
}

__global__ void
pyrDownKernel (const PtrStepSz<unsigned short> src, PtrStepSz<unsigned short> dst, float sigma_color)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  const int D = 5;

  int center = src.ptr (2 * y)[2 * x];

  int tx = min (2 * x - D / 2 + D, src.cols - 1);
  int ty = min (2 * y - D / 2 + D, src.rows - 1);
  int cy = max (0, 2 * y - D / 2);

  int sum = 0;
  int count = 0;

  for (; cy < ty; ++cy)
    for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
    {
      int val = src.ptr (cy)[cx];
      if (abs (val - center) < 3 * sigma_color)
      {
        sum += val;
        ++count;
      }
    }
  dst.ptr (y)[x] = sum / count;
}

__global__ void
pyrDownKernelIntensityGauss(const PtrStepSz<unsigned char> src, PtrStepSz<unsigned char> dst, float * gaussKernel)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  const int D = 5;

  int center = src.ptr (2 * y)[2 * x];

  int tx = min (2 * x - D / 2 + D, src.cols - 1);
  int ty = min (2 * y - D / 2 + D, src.rows - 1);
  int cy = max (0, 2 * y - D / 2);

  float sum = 0;
  int count = 0;

  for (; cy < ty; ++cy)
    for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
    {
        sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
        count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
    }
  dst.ptr (y)[x] = (sum / (float)count);
}

__global__ void
pyrDownKernelGaussF(const PtrStepSz<float> src, PtrStepSz<float> dst, float * gaussKernel)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  const int D = 5;

  float center = src.ptr (2 * y)[2 * x];

  int tx = min (2 * x - D / 2 + D, src.cols - 1);
  int ty = min (2 * y - D / 2 + D, src.rows - 1);
  int cy = max (0, 2 * y - D / 2);

  float sum = 0;
  int count = 0;

  for (; cy < ty; ++cy)
  {
      for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
      {
          if(!isnan(src.ptr (cy)[cx]))
          {
              sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
              count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
          }
      }
  }
  dst.ptr (y)[x] = (float)(sum / (float)count);
}

__global__ void
short2FloatKernel(const PtrStepSz<unsigned short> src, PtrStepSz<float> dst, int cutOff)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  dst.ptr(y)[x] = src.ptr(y)[x] > cutOff  || src.ptr(y)[x] <= 0 ? numeric_limits<float>::quiet_NaN() : ((float)src.ptr(y)[x]) / 1000.0f;
}

__global__ void
bgr2IntensityKernel(const PtrStepSz<PixelRGB> src, PtrStepSz<unsigned char> dst)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= dst.cols || y >= dst.rows)
    return;

  int value = (float)src.ptr(y)[x].r * 0.114f + (float)src.ptr(y)[x].b * 0.299f + (float)src.ptr(y)[x].g * 0.587f;

  dst.ptr (y)[x] = value;
}

__global__ void
truncateDepthKernel(PtrStepSz<unsigned short> depth, unsigned short max_distance_mm)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < depth.cols && y < depth.rows)
        if(depth.ptr(y)[x] > max_distance_mm)
            depth.ptr(y)[x] = 0;
}

__constant__ float gsobel_x3x3[9];
__constant__ float gsobel_y3x3[9];

template <int BLOCK_SIZE_X, int BLOCK_SIZE_Y, int PIXELS_PER_THREAD, int N, int N2>
__global__ void SGSobel_ke(const unsigned char* input_data,
                           unsigned short height,
                           unsigned short width,
                           unsigned short input_pitch,
                           unsigned short output_pitch,
                           short* output_dx,
                           short* output_dy) {

    short j = (blockIdx.x * BLOCK_SIZE_X) + threadIdx.x;
    short j0 = (blockIdx.x * BLOCK_SIZE_X * PIXELS_PER_THREAD) + (threadIdx.x * PIXELS_PER_THREAD);
    short i = (blockIdx.y * BLOCK_SIZE_Y) + threadIdx.y;

    unsigned int *ptr_ui;
    short *ptr_output_data;

    //Alloc and init shared memory
    __shared__ unsigned int input_data_smem[BLOCK_SIZE_Y+(N<<1)][BLOCK_SIZE_X+(N2<<1)];
    unsigned char *ptr_smem = (unsigned char*) &(input_data_smem[0][0]);
    unsigned short smem_pitch =  (BLOCK_SIZE_X+(N2<<1)) << 2;

    __shared__ float output_dx_smem[BLOCK_SIZE_Y][BLOCK_SIZE_X*PIXELS_PER_THREAD];
    __shared__ float output_dy_smem[BLOCK_SIZE_Y][BLOCK_SIZE_X*PIXELS_PER_THREAD];

#pragma unroll
    for(short p=0; p<PIXELS_PER_THREAD; p++){
        output_dx_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)+p] = 0;
        output_dy_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)+p] = 0;
    }

    if (i<height && j<(width>>2)) { //Assume PIXELS_PER_THREAD = 4

        //Each thread loads 1 uint, ie 4 uchar

        //Copy data to shared memory ----------------------------------------------------------------------------

        //1. All threads read, shift up and left
        ptr_ui = (unsigned int*) (input_data + ((i - N) * input_pitch));
        input_data_smem[threadIdx.y][threadIdx.x] = ptr_ui[j-N2];
        //2. Right columns
        if (threadIdx.x < (N2<<1)) {
            input_data_smem[threadIdx.y][threadIdx.x+BLOCK_SIZE_X] = (i-N>=0 && j-N2+BLOCK_SIZE_X<(width>>2)) ? ptr_ui[j-N2+BLOCK_SIZE_X] : 0;
        }
        //3. Bottom rows
        if (threadIdx.y < (N<<1)) {
            ptr_ui = (unsigned int*) (input_data + ((i - N + BLOCK_SIZE_Y) * input_pitch));
            input_data_smem[threadIdx.y+BLOCK_SIZE_Y][threadIdx.x] = (i-N+BLOCK_SIZE_Y<height && j-N2>=0) ? ptr_ui[j-N2] : 0;
        }
        //4. Bottom-right
        if (threadIdx.x < (N2<<1) && threadIdx.y < (N<<1)) {
            input_data_smem[threadIdx.y+BLOCK_SIZE_Y][threadIdx.x+BLOCK_SIZE_X] = (i-N+BLOCK_SIZE_Y<height && j-N2+BLOCK_SIZE_X<(width>>2)) ? ptr_ui[j-N2+BLOCK_SIZE_X] : 0;
        }
        __syncthreads();
        //-------------------------------------------------------------------------------------------------------

        //Processing --------------------------------------------------------------------------------------------
        short li = threadIdx.y + N;
        short lj = ((threadIdx.x + N2) * PIXELS_PER_THREAD);

        //3x3 neighbours
        short k=-N, l=-N;
#pragma unroll
        for(short loop = 0; loop < ((N<<1)+1)*((N<<1)+1); loop++) {

            short lik = li+k;
            short ljl = lj+l;

            //Get neighbour value
            unsigned char *ptr2 = ptr_smem + (lik * smem_pitch);

            int idx = ((N<<1)+1)*((N<<1)+1)-1-loop;
            float factor_x = gsobel_x3x3[idx];
            float factor_y = gsobel_y3x3[idx];

#pragma unroll
            for(short p=0; p<PIXELS_PER_THREAD; p++){

                //Get current_pixel value
                //	unsigned char val0 = ptr[lj+p];

                float valn = (float) ptr2[ljl+p];
                output_dx_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)+p] += factor_x *valn;
                output_dy_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)+p] += factor_y *valn;

            }//end for p

            l = (l<N) ? l+1 : -N;
            k = (l==-N) ? k+1 : k;
        }//end loop k,l

        __syncthreads();
        //-------------------------------------------------------------------------------------------------------

#pragma unroll
        for(short p=0; p<PIXELS_PER_THREAD; p++){
            ptr_output_data = (short*) ((unsigned char *) output_dx + (i * output_pitch));
            ptr_output_data[j0+p] = (short) output_dx_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)+p];
            ptr_output_data = (short*) ((unsigned char *) output_dy + (i * output_pitch));
            ptr_output_data[j0+p] = (short) output_dy_smem[threadIdx.y][(threadIdx.x * PIXELS_PER_THREAD)+p];
        }
    } //end if
}

int GetGridDim(int D, int B)
{
   return (D%B>0)?(D+(B-(D%B)))/B:D/B;
}

void sobelGaussian(DeviceArray2D<unsigned char>& src, DeviceArray2D<short>& dx, DeviceArray2D<short>& dy)
{
	float gsx3x3[9] = {0.52201,  0.00000, -0.52201,
			           0.79451, -0.00000, -0.79451,
			           0.52201,  0.00000, -0.52201};

	float gsy3x3[9] = {0.52201, 0.79451, 0.52201,
			           0.00000, 0.00000, 0.00000,
			           -0.52201, -0.79451, -0.52201};

	hipMemcpyToSymbol(HIP_SYMBOL(gsobel_x3x3), gsx3x3, 9<<2);
	hipMemcpyToSymbol(HIP_SYMBOL(gsobel_y3x3), gsy3x3, 9<<2);

	cudaSafeCall ( hipGetLastError () );
	cudaSafeCall (hipDeviceSynchronize ());

	SGSobel_ke<32, 6, 4, 1, 1><<<dim3(GetGridDim(dx.cols() / 4, 32), GetGridDim(dx.rows(), 6)), dim3(32, 6)>>>(src.ptr(0),
                                                                                                                (unsigned short) src.rows(),
                                                                                                                (unsigned short) src.cols(),
                                                                                                                (unsigned short) src.step(),
                                                                                                                (unsigned short) dx.step(),
                                                                                                                dx.ptr(0),
                                                                                                                dy.ptr(0));

	cudaSafeCall ( hipGetLastError () );
	cudaSafeCall (hipDeviceSynchronize ());
}

void
bilateralFilter (const DeviceArray2D<unsigned short>& src, DeviceArray2D<unsigned short>& dst)
{
  dim3 block (32, 8);
  dim3 grid (divUp (src.cols (), block.x), divUp (src.rows (), block.y));

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(bilateralKernel), hipFuncCachePreferL1);
  bilateralKernel<<<grid, block>>>(src, dst, 0.5f / (sigma_space * sigma_space), 0.5f / (sigma_color * sigma_color));

  cudaSafeCall ( hipGetLastError () );
};

void
pyrDown (const DeviceArray2D<unsigned short>& src, DeviceArray2D<unsigned short>& dst)
{
  dst.create (src.rows () / 2, src.cols () / 2);

  dim3 block (32, 8);
  dim3 grid (divUp (dst.cols (), block.x), divUp (dst.rows (), block.y));

  pyrDownGaussKernel<<<grid, block>>>(src, dst, sigma_color);
  cudaSafeCall ( hipGetLastError () );
};

void pyrDownGaussF(const DeviceArray2D<float>& src, DeviceArray2D<float> & dst)
{
  dst.create (src.rows () / 2, src.cols () / 2);

  dim3 block (32, 8);
  dim3 grid (divUp (dst.cols (), block.x), divUp (dst.rows (), block.y));

  const float gaussKernel[25] = {1, 4, 6, 4, 1,
  								 4, 16, 24, 16, 4,
  								 6, 24, 36, 24, 6,
  								 4, 16, 24, 16, 4,
  								 1, 4, 6, 4, 1};

  float * gauss_cuda;

  hipMalloc((void**) &gauss_cuda, sizeof(float) * 25);
  hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

  pyrDownKernelGaussF<<<grid, block>>>(src, dst, gauss_cuda);
  cudaSafeCall ( hipGetLastError () );

  hipFree(gauss_cuda);
};

void pyrDownUcharGauss(const DeviceArray2D<unsigned char>& src, DeviceArray2D<unsigned char> & dst)
{
  dst.create (src.rows () / 2, src.cols () / 2);

  dim3 block (32, 8);
  dim3 grid (divUp (dst.cols (), block.x), divUp (dst.rows (), block.y));

  const float gaussKernel[25] = {1, 4, 6, 4, 1,
  								 4, 16, 24, 16, 4,
  								 6, 24, 36, 24, 6,
  								 4, 16, 24, 16, 4,
  								 1, 4, 6, 4, 1};

  float * gauss_cuda;

  hipMalloc((void**) &gauss_cuda, sizeof(float) * 25);
  hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

  pyrDownKernelIntensityGauss<<<grid, block>>>(src, dst, gauss_cuda);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
  hipFree(gauss_cuda);
};

void shortDepthToMetres(const DeviceArray2D<unsigned short>& src, DeviceArray2D<float> & dst, int cutOff)
{
  dim3 block (32, 8);
  dim3 grid (divUp (dst.cols (), block.x), divUp (dst.rows (), block.y));

  short2FloatKernel<<<grid, block>>>(src, dst, cutOff);
  cudaSafeCall ( hipGetLastError () );
};

void imageBGRToIntensity(const DeviceArray2D<PixelRGB> & src, DeviceArray2D<unsigned char> & dst)
{
  dim3 block (32, 8);
  dim3 grid (divUp (dst.cols (), block.x), divUp (dst.rows (), block.y));

  bgr2IntensityKernel<<<grid, block>>>(src, dst);
  cudaSafeCall ( hipGetLastError () );
};
