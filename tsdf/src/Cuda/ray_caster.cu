#include "hip/hip_runtime.h"
// ray caster algorithm

#include "device.hpp"

__device__ __forceinline__ float
getMinTime (const float3& volume_max, const float3& origin, const float3& dir)
{
  float txmin = ( (dir.x > 0 ? 0.f : volume_max.x) - origin.x) / dir.x;
  float tymin = ( (dir.y > 0 ? 0.f : volume_max.y) - origin.y) / dir.y;
  float tzmin = ( (dir.z > 0 ? 0.f : volume_max.z) - origin.z) / dir.z;

  return fmax ( fmax (txmin, tymin), tzmin);
}

__device__ __forceinline__ float
getMaxTime (const float3& volume_max, const float3& origin, const float3& dir)
{
  float txmax = ( (dir.x > 0 ? volume_max.x : 0.f) - origin.x) / dir.x;
  float tymax = ( (dir.y > 0 ? volume_max.y : 0.f) - origin.y) / dir.y;
  float tzmax = ( (dir.z > 0 ? volume_max.z : 0.f) - origin.z) / dir.z;

  return fmin (fmin (txmax, tymax), tzmax);
}

struct RayCaster
{
  enum { CTA_SIZE_X = 32, CTA_SIZE_Y = 8 };

  Mat33 Rcurr;
  float3 tcurr;

  float time_step;
  float3 volume_size;

  float3 cell_size;
  int cols, rows;

  PtrStep<short> volume;

  Intr intr;

  mutable PtrStep<float> nmap;
  mutable PtrStep<float> vmap;

  int3 voxelWrap;

  mutable PtrStep<uchar4> vmap_curr_color;
  PtrStep<uchar4> color_volume;

  __device__ __forceinline__ float3
  get_ray_next (int x, int y) const
  {
    float3 ray_next;
    ray_next.x = (x - intr.cx) / intr.fx;
    ray_next.y = (y - intr.cy) / intr.fy;
    ray_next.z = 1;
    return ray_next;
  }

  __device__ __forceinline__ bool
  checkInds (const int3& g) const
  {
    return (g.x >= 0 && g.y >= 0 && g.z >= 0 && g.x < VOLUME_X && g.y < VOLUME_Y && g.z < VOLUME_X);
  }

  __device__ __forceinline__ float
  readTsdf (int x, int y, int z) const
  {
      const short * pos = &volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return unpack_tsdf (*pos);
  }

  __device__ __forceinline__ float
  readHeat (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->w;
  }

  __device__ __forceinline__ float
  readRed (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->x;
  }

  __device__ __forceinline__ float
  readGreen (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->y;
  }

  __device__ __forceinline__ float
  readBlue (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->z;
  }

  __device__ __forceinline__ int3
  getVoxel (float3 point) const
  {
    int vx = __float2int_rd (point.x / cell_size.x);        // round to negative infinity
    int vy = __float2int_rd (point.y / cell_size.y);
    int vz = __float2int_rd (point.z / cell_size.z);

    return make_int3 (vx, vy, vz);
  }

  // 3d linear interpolation
  __device__ __forceinline__ float
  interpolateTrilineary (const float3& origin, const float3& dir, float time) const
  {
    return interpolateTrilineary (origin + dir * time);
  }

  __device__ __forceinline__ float
  interpolateTrilineary (const float3& point) const
  {
    int3 g = getVoxel (point);

    if (g.x <= 0 || g.x >= VOLUME_X - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.y <= 0 || g.y >= VOLUME_Y - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.z <= 0 || g.z >= VOLUME_Z - 1)
      return numeric_limits<float>::quiet_NaN ();

    float vx = (g.x + 0.5f) * cell_size.x;
    float vy = (g.y + 0.5f) * cell_size.y;
    float vz = (g.z + 0.5f) * cell_size.z;

    g.x = (point.x < vx) ? (g.x - 1) : g.x;
    g.y = (point.y < vy) ? (g.y - 1) : g.y;
    g.z = (point.z < vz) ? (g.z - 1) : g.z;

    float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
    float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
    float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

    float res = readTsdf (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                readTsdf (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                readTsdf (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                readTsdf (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                readTsdf (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                readTsdf (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                readTsdf (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                readTsdf (g.x + 1, g.y + 1, g.z + 1) * a * b * c;
    return res;
  }

  __device__ __forceinline__ uchar3
  interpolateColorTrilineary (const float3& point) const
  {
    int3 g = getVoxel (point);

    uchar3 black = {0, 0, 0};

    if (g.x <= 0 || g.x >= VOLUME_X - 1)
      return black;

    if (g.y <= 0 || g.y >= VOLUME_Y - 1)
      return black;

    if (g.z <= 0 || g.z >= VOLUME_Z - 1)
      return black;

    float vx = (g.x + 0.5f) * cell_size.x;
    float vy = (g.y + 0.5f) * cell_size.y;
    float vz = (g.z + 0.5f) * cell_size.z;

    g.x = (point.x < vx) ? (g.x - 1) : g.x;
    g.y = (point.y < vy) ? (g.y - 1) : g.y;
    g.z = (point.z < vz) ? (g.z - 1) : g.z;

    float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
    float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
    float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

    uchar3 res = {readRed (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                  readRed (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                  readRed (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                  readRed (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                  readRed (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                  readRed (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                  readRed (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                  readRed (g.x + 1, g.y + 1, g.z + 1) * a * b * c,
                  readGreen (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                  readGreen (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                  readGreen (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                  readGreen (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                  readGreen (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                  readGreen (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                  readGreen (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                  readGreen (g.x + 1, g.y + 1, g.z + 1) * a * b * c,
                  readBlue (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                  readBlue (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                  readBlue (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                  readBlue (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                  readBlue (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                  readBlue (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                  readBlue (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                  readBlue (g.x + 1, g.y + 1, g.z + 1) * a * b * c};

    return res;
  }

  __device__ __forceinline__ float
  interpolateHeatTrilineary (const float3& point) const
  {
    int3 g = getVoxel (point);

    if (g.x <= 0 || g.x >= VOLUME_X - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.y <= 0 || g.y >= VOLUME_Y - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.z <= 0 || g.z >= VOLUME_Z - 1)
      return numeric_limits<float>::quiet_NaN ();

    float vx = (g.x + 0.5f) * cell_size.x;
    float vy = (g.y + 0.5f) * cell_size.y;
    float vz = (g.z + 0.5f) * cell_size.z;

    g.x = (point.x < vx) ? (g.x - 1) : g.x;
    g.y = (point.y < vy) ? (g.y - 1) : g.y;
    g.z = (point.z < vz) ? (g.z - 1) : g.z;

    float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
    float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
    float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

    float res = readHeat (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                readHeat (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                readHeat (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                readHeat (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                readHeat (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                readHeat (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                readHeat (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                readHeat (g.x + 1, g.y + 1, g.z + 1) * a * b * c;
    return res;
  }

  __device__ __forceinline__ void
  operator () () const
  {
    int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
    int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

    if (x >= cols || y >= rows)
      return;

    vmap.ptr (y)[x] = numeric_limits<float>::quiet_NaN ();
    nmap.ptr (y)[x] = numeric_limits<float>::quiet_NaN ();

    float3 ray_start = tcurr;
    float3 ray_next = Rcurr * get_ray_next (x, y) + tcurr;

    float3 ray_dir = normalized (ray_next - ray_start);

    //ensure that it isn't a degenerate case
    ray_dir.x = (ray_dir.x == 0.f) ? 1e-15 : ray_dir.x;
    ray_dir.y = (ray_dir.y == 0.f) ? 1e-15 : ray_dir.y;
    ray_dir.z = (ray_dir.z == 0.f) ? 1e-15 : ray_dir.z;

    // computer time when entry and exit volume: ray = ray_start + t * ray_dir enters and exits the volume [0,0,0] * [VX,VY,VZ]
    float time_start_volume = getMinTime (volume_size, ray_start, ray_dir);
    float time_exit_volume = getMaxTime (volume_size, ray_start, ray_dir);

    const float min_dist = 0.f;         //in meters
	// make the time start volume positive
    time_start_volume = fmax (time_start_volume, min_dist);
	// if some error occurs, return
    if (time_start_volume >= time_exit_volume)
      return;

	// get the voxel position of the first ray position
    float time_curr = time_start_volume;
    int3 g = getVoxel (ray_start + ray_dir * time_curr);
    g.x = max (0, min (g.x, VOLUME_X - 1));
    g.y = max (0, min (g.y, VOLUME_Y - 1));
    g.z = max (0, min (g.z, VOLUME_Z - 1));

	// get the first tsdf value
    float tsdf = readTsdf (g.x, g.y, g.z);

    //infinite loop guard
    const float max_time = 3 * (volume_size.x + volume_size.y + volume_size.z);

    for (; time_curr < max_time; time_curr += time_step)
    {
      float tsdf_prev = tsdf;

	  // move to next ray position (time step = trunc dist * 0.8)
      int3 g = getVoxel (  ray_start + ray_dir * (time_curr + time_step)  );
      if (!checkInds (g))
        break;

	  // get the tsdf value
      tsdf = readTsdf (g.x, g.y, g.z);

	  // we want to find the crossing point, this state means that we are in a wrong viewing position or some stange error has occured
      if (tsdf_prev < 0.f && tsdf > 0.f)
        break;

      if (tsdf_prev > 0.f && tsdf < 0.f)           //zero crossing
      {
	    // refine the current tsdf value using trilinear interpolation
        float Ftdt = interpolateTrilineary (ray_start, ray_dir, time_curr + time_step);
        if (isnan (Ftdt))
          break;

		// refine the previous tsdf value using trilinear interpolation
        float Ft = interpolateTrilineary (ray_start, ray_dir, time_curr);
        if (isnan (Ft))
          break;

		// get the sub pixel optimal zero crossing point assuming linear model
        float Ts = time_curr - time_step * Ft / (Ftdt - Ft);

		// may be wrong 
        float3 vetex_found = ray_start + ray_dir * Ts;

		// found the crossing point
        vmap.ptr (y       )[x] = vetex_found.x;
        vmap.ptr (y + rows)[x] = vetex_found.y;
        vmap.ptr (y + 2 * rows)[x] = vetex_found.z;

        int3 g = getVoxel ( ray_start + ray_dir * time_curr );

		// get the point color and the heat weight
        uchar3 pointColor = interpolateColorTrilineary(vetex_found);

        vmap_curr_color.ptr(y)[x].x = pointColor.x;
        vmap_curr_color.ptr(y)[x].y = pointColor.y;
        vmap_curr_color.ptr(y)[x].z = pointColor.z;
        vmap_curr_color.ptr(y)[x].w = interpolateHeatTrilineary(vetex_found);

		// estimate the normal direction (dfdx, dfdy, dfdz)
        if (g.x > 1 && g.y > 1 && g.z > 1 && g.x < VOLUME_X - 2 && g.y < VOLUME_Y - 2 && g.z < VOLUME_Z - 2)
        {
          float3 t;
          float3 n;

          t = vetex_found;
          t.x += cell_size.x;
          float Fx1 = interpolateTrilineary (t);

          t = vetex_found;
          t.x -= cell_size.x;
          float Fx2 = interpolateTrilineary (t);

          n.x = (Fx1 - Fx2);

          t = vetex_found;
          t.y += cell_size.y;
          float Fy1 = interpolateTrilineary (t);

          t = vetex_found;
          t.y -= cell_size.y;
          float Fy2 = interpolateTrilineary (t);

          n.y = (Fy1 - Fy2);

          t = vetex_found;
          t.z += cell_size.z;
          float Fz1 = interpolateTrilineary (t);

          t = vetex_found;
          t.z -= cell_size.z;
          float Fz2 = interpolateTrilineary (t);

          n.z = (Fz1 - Fz2);

          n = normalized (n);

          nmap.ptr (y       )[x] = n.x;
          nmap.ptr (y + rows)[x] = n.y;
          nmap.ptr (y + 2 * rows)[x] = n.z;
        }
        break;
      }
    }          /* for(;;)  */
  }
};

__global__ void
rayCastKernel (const RayCaster rc) {
  rc ();
}

void
raycast (const Intr& intr, const Mat33& Rcurr, const float3& tcurr, 
                      float tranc_dist, const float3& volume_size,
                      const PtrStep<short>& volume, DeviceArray2D<float>& vmap, DeviceArray2D<float>& nmap, const int3 & voxelWrap, DeviceArray2D<uchar4> & vmap_curr_color,
                      PtrStep<uchar4> color_volume)
{
  RayCaster rc;

  rc.Rcurr = Rcurr;
  rc.tcurr = tcurr;

  rc.time_step = tranc_dist * 0.8f;

  rc.volume_size = volume_size;

  rc.cell_size.x = volume_size.x / VOLUME_X;
  rc.cell_size.y = volume_size.y / VOLUME_Y;
  rc.cell_size.z = volume_size.z / VOLUME_Z;

  rc.cols = vmap.cols ();
  rc.rows = vmap.rows () / 3;

  rc.intr = intr;

  rc.volume = volume;
  rc.vmap = vmap;
  rc.nmap = nmap;

  rc.voxelWrap = voxelWrap;

  rc.vmap_curr_color = vmap_curr_color;
  rc.color_volume = color_volume;

  // ray cast as a new depth camera capture
  dim3 block (RayCaster::CTA_SIZE_X, RayCaster::CTA_SIZE_Y);
  dim3 grid (divUp (rc.cols, block.x), divUp (rc.rows, block.y));

  rayCastKernel<<<grid, block>>>(rc);
  cudaSafeCall (hipGetLastError ());
}

